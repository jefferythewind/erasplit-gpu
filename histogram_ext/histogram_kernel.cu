#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define F_TILE 128  // Number of features processed per block (tile)

// Each block processes a tile of features (of size up to F_TILE) and a chunk of samples.
__global__ void histogram_kernel_shared_sample(
    const int8_t* __restrict__ bin_indices, // [N, F] bin indices
    const float* __restrict__ gradients,      // [N] gradient values
    float* __restrict__ grad_hist,            // [F * B] global gradient histogram (flattened)
    float* __restrict__ hess_hist,            // [F * B] global hessian histogram (flattened)
    int64_t N, int64_t F, int64_t B
) {
    // Use dynamic shared memory to hold the histogram for a tile.
    // Allocate 2 arrays: one for gradients and one for hessians.
    extern __shared__ float shmem[];
    float* shared_grad = shmem;                     // size: tile_features * B floats
    float* shared_hess = shmem + (F_TILE * B);        // same size

    int tid = threadIdx.x;             // Use a 1D block (for sample processing)
    int block_size = blockDim.x;

    // Each block is assigned a tile of features:
    int feature_offset = blockIdx.x * F_TILE;
    // Adjust tile width if we're near the end of the feature dimension.
    int tile_features = (feature_offset + F_TILE > F) ? (F - feature_offset) : F_TILE;
    int tile_size = tile_features * B; // total number of bins in this feature tile

    // Initialize the tile’s shared memory histograms.
    for (int i = tid; i < tile_size; i += block_size) {
        shared_grad[i] = 0.0f;
        shared_hess[i] = 0.0f;
    }
    __syncthreads();

    // Each block also covers a chunk of samples. Determine the sample index
    int sample = blockIdx.y * block_size + tid;
    if (sample < N) {
        // For each feature in this tile, compute the bin and update shared histograms.
        for (int j = 0; j < tile_features; j++) {
            // Global feature index.
            int f_idx = feature_offset + j;
            int64_t idx = sample * F + f_idx;  // index into the [N, F] bin_indices tensor
            int8_t b = bin_indices[idx];       // get bin index
            if (b >= 0 && b < B) {
                int shared_idx = j * B + b;    // index into the tile histogram in shared memory
                // Using atomics because several threads may update the same bin.
                atomicAdd(&shared_grad[shared_idx], gradients[sample]);
                atomicAdd(&shared_hess[shared_idx], 1.0f);
            }
        }
    }
    __syncthreads();

    // Flush the per-tile histograms from shared memory to global memory.
    // Each bin in the tile is added to the global histogram (which is sized [F, B]).
    for (int i = tid; i < tile_size; i += block_size) {
        int local_feature = i / B; // feature index relative to the tile
        int bin = i % B;           // bin index
        int f_idx = feature_offset + local_feature;
        if (f_idx < F) {
            int global_idx = f_idx * B + bin;
            atomicAdd(&grad_hist[global_idx], shared_grad[i]);
            atomicAdd(&hess_hist[global_idx], shared_hess[i]);
        }
    }
}

void launch_histogram_kernel_cuda(
    const at::Tensor& bin_indices,   // [N, F] int8 tensor
    const at::Tensor& gradients,       // [N] float tensor
    at::Tensor& grad_hist,             // [F * B] float tensor (preallocated)
    at::Tensor& hess_hist,             // [F * B] float tensor (preallocated)
    int num_bins                 // B (number of bins)
) {
    int64_t N = bin_indices.size(0);
    int64_t F = bin_indices.size(1);
    int64_t B = num_bins;

    // Define grid and block dimensions.
    // blockDim.x: number of threads per block (for processing samples).
    int threads_per_block = 256;
    // gridDim.x: number of feature tiles.
    int grid_x = (F + F_TILE - 1) / F_TILE;
    // gridDim.y: number of sample chunks.
    int grid_y = (N + threads_per_block - 1) / threads_per_block;
    dim3 blocks(grid_x, grid_y);
    dim3 threads(threads_per_block);

    // Calculate shared memory size:
    // We allocate 2 arrays of size (F_TILE * B) floats (one for grad and one for hess).
    size_t shared_mem_size = 2 * F_TILE * B * sizeof(float);

    histogram_kernel_shared_sample<<<blocks, threads, shared_mem_size>>>(
        bin_indices.data_ptr<int8_t>(),
        gradients.data_ptr<float>(),
        grad_hist.data_ptr<float>(),
        hess_hist.data_ptr<float>(),
        N, F, B
    );
}
